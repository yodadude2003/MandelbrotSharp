// Generated by Hybridizer version 1.0.0.0
 #include "hip/hip_runtime.h"                                                                                 
                                                                      
                                                                                                             
 #if defined(__HIPCC__)                                                                                     
 	#ifndef hyb_device                                                                                       
 		#define hyb_inline __forceinline__                                                                   
 		                                                                                                     
 		#define hyb_constant __constant__                                                                    
 		#if defined(HYBRIDIZER_NO_HOST)                                                                      
 			#define hyb_host                                                                                 
 			#define	hyb_device  __device__                                                                   
 		#else                                                                                                
 			#define hyb_host __host__                                                                        
 			#define	hyb_device  __device__                                                                   
 		#endif                                                                                               
 	#endif                                                                                                   
 #else                                                                                                        
 	#ifndef hyb_device                                                                                       
 		#define hyb_inline inline                                                                            
 		#define hyb_device                                                                                   
 		#define hyb_constant                                                                                 
 	#endif                                                                                                   
 #endif                                                                                                       
                                                                                                              
 #pragma once                                                                                                 
 #if defined _WIN32 || defined _WIN64 || defined __CYGWIN__                                                   
   #define BUILDING_DLL                                                                                       
   #ifdef BUILDING_DLL                                                                                        
     #ifdef __GNUC__                                                                                          
       #define DLL_PUBLIC __attribute__ ((dllexport))                                                         
     #else                                                                                                    
       #define DLL_PUBLIC __declspec(dllexport) // Note: actually gcc seems to also supports this syntax.     
     #endif                                                                                                   
   #else                                                                                                      
     #ifdef __GNUC__                                                                                          
       #define DLL_PUBLIC __attribute__ ((dllimport))                                                         
     #else                                                                                                    
       #define DLL_PUBLIC __declspec(dllimport) // Note: actually gcc seems to also supports this syntax.     
     #endif                                                                                                   
   #endif                                                                                                     
   #define DLL_LOCAL                                                                                          
 #else                                                                                                        
   #if __GNUC__ >= 4                                                                                          
     #define DLL_PUBLIC __attribute__ ((visibility ("default")))                                            
     #define DLL_LOCAL  __attribute__ ((visibility ("hidden")))                                             
   #else                                                                                                      
     #define DLL_PUBLIC                                                                                       
     #define DLL_LOCAL                                                                                        
   #endif                                                                                                     
 #endif                                                                                                       


// hybridizer core types
#include <cstdint>
namespace hybridizer { struct hybridobject ; }
namespace hybridizer { struct runtime ; }

#pragma region defined enums and types
#if defined(__cplusplus) || defined(__HIPCC__)
namespace MandelBrot { 
struct GPUFractal ;
} // Leaving namespace
namespace System { 
struct Math ;
} // Leaving namespace
#endif // TOTO
#pragma endregion

extern "C" void* __hybridizer_init_basic_runtime();
#include <cstdio>
// generating GetTypeID function
#include <cstring> // for strcmp
extern "C" DLL_PUBLIC int HybridizerGetTypeID( const char* fullTypeName)
{
	if (strcmp (fullTypeName, "Hybridizer.Runtime.CUDAImports.blockDim") == 0) return 1000000 ; 
	if (strcmp (fullTypeName, "Hybridizer.Runtime.CUDAImports.blockIdx") == 0) return 1000001 ; 
	if (strcmp (fullTypeName, "Hybridizer.Runtime.CUDAImports.gridDim") == 0) return 1000002 ; 
	if (strcmp (fullTypeName, "Hybridizer.Runtime.CUDAImports.threadIdx") == 0) return 1000003 ; 
	if (strcmp (fullTypeName, "MandelBrot.GPUFractal") == 0) return 1000004 ; 
	if (strcmp (fullTypeName, "System.Math") == 0) return 1000005 ; 
	return 0 ;
}
extern "C" DLL_PUBLIC const char* HybridizerGetTypeFromID( const int typeId)
{
	if (typeId == 1000000) return "Hybridizer.Runtime.CUDAImports.blockDim" ; 
	if (typeId == 1000001) return "Hybridizer.Runtime.CUDAImports.blockIdx" ; 
	if (typeId == 1000002) return "Hybridizer.Runtime.CUDAImports.gridDim" ; 
	if (typeId == 1000003) return "Hybridizer.Runtime.CUDAImports.threadIdx" ; 
	if (typeId == 1000004) return "MandelBrot.GPUFractal" ; 
	if (typeId == 1000005) return "System.Math" ; 
	return "" ;
}
extern "C" DLL_PUBLIC int HybridizerGetShallowSize (const char* fullTypeName) 
{
	#ifdef __TYPE_DECL__MandelBrot_GPUFractal___
	if (strcmp (fullTypeName, "MandelBrot.GPUFractal") == 0) return 8 ; 
	#endif
	return 0 ;
}

// Get various Hybridizer properties at runtime
struct __hybridizer_properties {
    int32_t UseHybridArrays;
    int32_t Flavor;
    int32_t CompatibilityMode;
    int32_t _dummy;
};
extern "C" DLL_PUBLIC __hybridizer_properties __HybridizerGetProperties () {
    __hybridizer_properties res;
    res.UseHybridArrays = 0;
    res.Flavor = 1;
    res.CompatibilityMode = 0;
    return res ;
}
                                     
 struct HybridModule                                  
 {                                                    
     void* module_data ;                              
     hipModule_t module ;                                
 } ;                                                  
                                                      
 extern char __hybridizer_cubin_module_data [] ;      
 static HybridModule __hybridizer__gs_module = { 0 }; 

#pragma region Wrappers definitions


extern "C" DLL_PUBLIC int run_ExternCWrapper_CUDA( int gridDim_x,  int gridDim_y,  int blockDim_x,  int blockDim_y,  int blockDim_z,  int shared,  double* const data_in,  int lineFrom,  int lineTo,  int N,  int M,  int frameNum,  int maxiter)
{
	hipError_t cures ;                                                                                 
	if (__hybridizer__gs_module.module_data == 0)                                                    
	{                                                                                              
		cures = hipModuleLoadData (&(__hybridizer__gs_module.module), __hybridizer_cubin_module_data) ; 
		if (cures != hipSuccess) return (int)cures ;                                                 
	}                                                                                              
	                                                                                                 
	hipFunction_t __hybridizer__cufunc ;                                                                
	                                                                                                 
	cures = hipModuleGetFunction (&__hybridizer__cufunc, __hybridizer__gs_module.module, "run") ;   
	if (cures != hipSuccess) return (int)cures ;                                                   
	                                                                                                 
	hybridizer::runtime* __hybridizer_runtime = (hybridizer::runtime*) __hybridizer_init_basic_runtime(); 



	void* __hybridizer_launch_config[9] = 
		{
			(void*)&__hybridizer_runtime,
			(void*)&data_in,
			(void*)&lineFrom,
			(void*)&lineTo,
			(void*)&N,
			(void*)&M,
			(void*)&frameNum,
			(void*)&maxiter,
			(void*)0
		} ;

	shared += 16 ; if (shared > 48*1024) shared = 48*1024 ;                                                                                                
	                                                                                                                                                       
	cures = hipModuleLaunchKernel (__hybridizer__cufunc, gridDim_x, gridDim_y, 1, blockDim_x, blockDim_y, blockDim_z, shared, 0, __hybridizer_launch_config, 0) ; 
	if (cures != hipSuccess) return (int)cures ; 
	int cudaLaunchRes = (int)::hipPeekAtLastError ();                                                                                                     
	if (cudaLaunchRes != 0) return cudaLaunchRes;                                                                                                          
	int __synchronizeRes = (int)::hipDeviceSynchronize () ;                                                                                               
	return __synchronizeRes ;                                                                                                                              

}

#pragma endregion
